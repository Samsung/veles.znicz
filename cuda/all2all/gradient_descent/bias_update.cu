#include "hip/hip_runtime.h"
#include "gradient_descent_common.cu"


/// @brief Calculate gradient for bias update.
/// @param bias Layer bias.
/// @param err_output Backpropagated error.
/// @param gradient Computed gradient.
/// @param gradient_with_moment Accumulated gradient with moments.
/// @param lr learning_rate.
/// @param factor_l12 lnorm_factor.
/// @param l1_vs_l2 how much to prefer l1 over l2 (from [0, 1]).
/// @param moment Moment for gradient.
/// @details Should be defined externally:
///          REDUCE_SIZE - size of the block for matrix reduce,
///          BIAS_SIZE - bias size (Y),
///          OUTPUT_SIZE - number of output elements in the minibatch (BATCH).
extern "C"
__global__ void bias_update(const dtype    *err_output,
                            dtype          *bias,
                            dtype          *gradient,
                            dtype          *accumulated_gradient,
                            dtype          *gradient_with_moment,
                            const dtype    lr,
                            const dtype    factor_l12,
                            const dtype    l1_vs_l2,
                            const dtype    moment) {

  #define A err_output
  #define A_WIDTH BIAS_SIZE
  #define A_HEIGHT OUTPUT_SIZE
  #define A_COL

  #include "matrix_reduce.cu"

  #undef A_COL
  #undef A_HEIGHT
  #undef A_WIDTH
  #undef A

  #include "bias_update.store_output.cu"
}
