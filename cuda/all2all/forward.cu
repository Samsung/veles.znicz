#include "hip/hip_runtime.h"
#ifndef OUTPUT_SAMPLE_SIZE
#error "OUTPUT_SAMPLE_SIZE must be defined"
#endif

extern "C"
__global__ void apply_bias_with_activation(dtype *output, const dtype *bias) {
  size_t idx = blockIdx.y * OUTPUT_SAMPLE_SIZE + blockIdx.x;
  dtype y = output[idx];
  #if INCLUDE_BIAS > 0
    y += bias[blockIdx.x];
  #endif
  #if ACTIVATION_LINEAR > 0
    output[idx] = y;
  #elif ACTIVATION_TANH > 0
    output[idx] = (dtype)1.7159 * tanh((dtype)0.6666 * y);
  #elif ACTIVATION_RELU > 0
    output[idx] = y > 15 ? y : log(exp(y) + 1);
  #elif ACTIVATION_STRICT_RELU > 0
    output[idx] = max(y, (dtype)0.0);
  #else
    #error "Unsupported activation"
  #endif
}
