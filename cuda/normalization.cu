#include "hip/hip_runtime.h"
// Local response normalization kernels for ReLU units.
// Detailed description given in article by Krizhevsky, Sutskever and Hinton:
// "ImageNet Classification with Deep Convolutional Neural Networks"
//

#include "defines.cu"
#include "highlight.cuh"

//#define ALPHA 0.0001
//#define BETA 0.75
//#define K 2
//#define N 3
//#define NUM_OF_CHANS 5

__device__ void calculate_subsums(const dtype *h, dtype *subsums) {
  for (int i = 0; i < NUM_OF_CHANS; i++) {
    subsums[i] = 0;
  }

  int min_index = 0;
  int max_index = min(N / 2, NUM_OF_CHANS - 1);
  for(int i = min_index; i <= max_index; i++) {
    subsums[0] += h[i] * h[i];
  }

  for (int i = 1; i < NUM_OF_CHANS; i++) {
    int new_min_index = max(0, i - N / 2);
    int new_max_index = min(i + N / 2, NUM_OF_CHANS - 1);
    dtype subsum = subsums[i - 1];

    for(int j = min_index; j < new_min_index; j++) {
      subsum -= h[j] * h[j];
    }

    for(int j = max_index + 1; j <= new_max_index; j++) {
      subsum += h[j] * h[j];
    }

    subsums[i] = subsum;
    min_index = new_min_index;
    max_index = new_max_index;
  }
}

extern "C"
__global__ void forward(const dtype *in_data, dtype *out_data) {
  int global_index = threadIdx.x + blockIdx.x * blockDim.x;
  if (global_index >= OUTPUT_SIZE) {
    return;
  }
  int global_offset = global_index * NUM_OF_CHANS;

  dtype h[NUM_OF_CHANS];
  for(int i = 0; i < NUM_OF_CHANS; i++) {
    h[i] = in_data[global_offset + i];
  }

  dtype subsums[NUM_OF_CHANS];
  calculate_subsums(h, subsums);

  for(int i = 0; i < NUM_OF_CHANS; i++) {
    out_data[global_offset + i] = in_data[global_offset + i] *
        pow((dtype)K + (dtype)ALPHA * subsums[i], (dtype)(-BETA));
  }
}

extern "C"
__global__ void backward(
    const dtype *in_err_y, const dtype *in_h, dtype *out_err_h) {
  int global_index = threadIdx.x + blockIdx.x * blockDim.x;
  if (global_index >= OUTPUT_SIZE) {
    return;
  }
  int global_offset = global_index * NUM_OF_CHANS;

  dtype h[NUM_OF_CHANS];
  for(int i = 0; i < NUM_OF_CHANS; i++) {
    h[i] = in_h[global_offset + i];
  }

  dtype subsums[NUM_OF_CHANS];
  calculate_subsums(h, subsums);

  for(int i = 0; i < NUM_OF_CHANS; i++) {
    subsums[i] = K + ALPHA * subsums[i];
  }

  dtype local_err_y[NUM_OF_CHANS];
  for(int i = 0; i < NUM_OF_CHANS; i++) {
    local_err_y[i] = in_err_y[global_offset + i];
  }

  for(int i = 0; i < NUM_OF_CHANS; i++) {
    dtype delta_h = 0;

    int min_index = max(0, i - N / 2);
    int max_index = min(i + N / 2, NUM_OF_CHANS - 1);

    for(int j = min_index; j <= max_index; j++) {
      dtype dh = 0;
      if(i == j) {
        dh += subsums[j];
      }
      dh -= 2 * ALPHA * BETA * h[i] * h[j];
      dh *= local_err_y[j] / pow(subsums[j], (dtype)(BETA + 1));
      delta_h += dh;
    }
    out_err_h[global_offset + i] = delta_h;
  }
}
